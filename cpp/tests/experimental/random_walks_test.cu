/*
 * Copyright (c) 2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

#include "hip/hip_runtime_api.h"
#include "gtest/gtest.h"

#include <utilities/base_fixture.hpp>
#include <utilities/test_utilities.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/random.h>

#include <algorithms.hpp>
#include <experimental/random_walks.cuh>
#include <graph.hpp>

#include <raft/handle.hpp>
#include <raft/random/rng.cuh>

#include <algorithm>
#include <iterator>
#include <limits>
#include <numeric>
#include <utilities/high_res_timer.hpp>
#include <vector>

struct RandomWalks_Usecase {
  std::string graph_file_full_path{};
  bool test_weighted{false};

  RandomWalks_Usecase(std::string const& graph_file_path, bool test_weighted)
    : test_weighted(test_weighted)
  {
    if ((graph_file_path.length() > 0) && (graph_file_path[0] != '/')) {
      graph_file_full_path = cugraph::test::get_rapids_dataset_root_dir() + "/" + graph_file_path;
    } else {
      graph_file_full_path = graph_file_path;
    }
  };
};

class Tests_RandomWalks : public ::testing::TestWithParam<RandomWalks_Usecase> {
 public:
  Tests_RandomWalks() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t, typename weight_t>
  void run_current_test(RandomWalks_Usecase const& configuration)
  {
    raft::handle_t handle{};

    std::cout << "read graph file: " << configuration.graph_file_full_path << std::endl;

    cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, false, false> graph(handle);
    std::tie(graph, std::ignore) =
      cugraph::test::read_graph_from_matrix_market_file<vertex_t, edge_t, weight_t, false, false>(
        handle, configuration.graph_file_full_path, configuration.test_weighted, false);

    auto graph_view = graph.view();
    uint64_t seed{101201301401};
    raft::random::Rng rnd(seed);  // TODO: have it passed as argument from the test instance;

    // call random_walks:
    start_random_walks(graph_view, rnd);
  }

  template <typename graph_t, typename random_eng_t>
  void start_random_walks(graph_t const& graph, random_eng_t& rnd_eng)
  {
    using vertex_t = typename graph_t::vertex_type;
    using weight_t = typename graph_t::weight_type;

    raft::handle_t handle{};
    std::vector<vertex_t> empty_vec{};  // for now...
    size_t max_d{10};

    auto ret_tuple =
      cugraph::experimental::detail::random_walks(handle, graph, empty_vec, max_d, rnd_eng);
  }
};

TEST_P(Tests_RandomWalks, Initialize_i32_i32_f)
{
  run_current_test<int32_t, int32_t, float>(GetParam());
}

INSTANTIATE_TEST_CASE_P(simple_test,
                        Tests_RandomWalks,
                        ::testing::Values(RandomWalks_Usecase("test/datasets/karate.mtx", true)
#if 0
			,
                                          RandomWalks_Usecase("test/datasets/web-Google.mtx", true),
                                          RandomWalks_Usecase("test/datasets/ljournal-2008.mtx", true),
                                          RandomWalks_Usecase("test/datasets/webbase-1M.mtx", true)
#endif
                                            ));

CUGRAPH_TEST_PROGRAM_MAIN()

/*
struct RandomWalksTest : public ::testing::Test {
};

TEST_F(RandomWalksTest, CorrectInit)
{
  raft::handle_t handle{};

  ASSERT_TRUE(true);
}
*/
