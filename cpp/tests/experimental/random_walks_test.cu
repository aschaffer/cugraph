/*
 * Copyright (c) 2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

#include "hip/hip_runtime_api.h"
#include "gtest/gtest.h"

#include <utilities/base_fixture.hpp>
#include <utilities/test_utilities.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/random.h>

#include <algorithms.hpp>
#include <experimental/random_walks.cuh>
#include <graph.hpp>

#include <raft/handle.hpp>
#include <raft/random/rng.cuh>

#include <algorithm>
#include <iterator>
#include <limits>
#include <numeric>
#include <utilities/high_res_timer.hpp>
#include <vector>

namespace {  // anonym.
template <typename vertex_t, typename index_t>
void fill_start(raft::handle_t const& handle,
                rmm::device_uvector<vertex_t>& d_start,
                index_t num_vertices)
{
  index_t num_paths = d_start.size();

  thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                    thrust::make_counting_iterator<index_t>(0),
                    thrust::make_counting_iterator<index_t>(num_paths),

                    d_start.begin(),
                    [num_vertices] __device__(auto indx) { return indx % num_vertices; });
}
}  // namespace

struct RandomWalks_Usecase {
  std::string graph_file_full_path{};
  bool test_weighted{false};

  RandomWalks_Usecase(std::string const& graph_file_path, bool test_weighted)
    : test_weighted(test_weighted)
  {
    if ((graph_file_path.length() > 0) && (graph_file_path[0] != '/')) {
      graph_file_full_path = cugraph::test::get_rapids_dataset_root_dir() + "/" + graph_file_path;
    } else {
      graph_file_full_path = graph_file_path;
    }
  };
};

class Tests_RandomWalks : public ::testing::TestWithParam<RandomWalks_Usecase> {
 public:
  Tests_RandomWalks() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t, typename weight_t>
  void run_current_test(RandomWalks_Usecase const& configuration)
  {
    raft::handle_t handle{};

    std::cout << "read graph file: " << configuration.graph_file_full_path << std::endl;

    cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, false, false> graph(handle);
    std::tie(graph, std::ignore) =
      cugraph::test::read_graph_from_matrix_market_file<vertex_t, edge_t, weight_t, false, false>(
        handle, configuration.graph_file_full_path, configuration.test_weighted, false);

    auto graph_view = graph.view();

    // call random_walks:
    start_random_walks(graph_view);
  }

  template <typename graph_t>
  void start_random_walks(graph_t const& graph)
  {
    using vertex_t = typename graph_t::vertex_type;
    using weight_t = typename graph_t::weight_type;

    raft::handle_t handle{};
    vertex_t num_paths = 10;
    rmm::device_uvector<vertex_t> d_start(num_paths, handle.get_stream());

    vertex_t num_vertices = graph.get_number_of_vertices();
    fill_start(handle, d_start, num_vertices);

    vertex_t max_d{10};

    auto ret_tuple = cugraph::experimental::random_walks(handle, graph, d_start, max_d);
  }
};

TEST_P(Tests_RandomWalks, Initialize_i32_i32_f)
{
  run_current_test<int32_t, int32_t, float>(GetParam());
}

INSTANTIATE_TEST_CASE_P(simple_test,
                        Tests_RandomWalks,
                        ::testing::Values(RandomWalks_Usecase("test/datasets/karate.mtx", true)
#if 0
			,
                                          RandomWalks_Usecase("test/datasets/web-Google.mtx", true),
                                          RandomWalks_Usecase("test/datasets/ljournal-2008.mtx", true),
                                          RandomWalks_Usecase("test/datasets/webbase-1M.mtx", true)
#endif
                                            ));

CUGRAPH_TEST_PROGRAM_MAIN()

/*
struct RandomWalksTest : public ::testing::Test {
};

TEST_F(RandomWalksTest, CorrectInit)
{
  raft::handle_t handle{};

  ASSERT_TRUE(true);
}
*/
